#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "parameter.h"
#include "tree.h"
#include "particle.h"

// Not optimized (too many global memory access)
__global__ void Particle_Cell_Kernel(float4* P, int ng, float4* C, int nl, float3* F, float epsilon) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int ip = id % ng;
    float3 f = {0.0, 0.0, 0.0};
    float4 p = P[ip];
    while(id < ng * (nl + ng)){
        // Particle-Cell
        if(id < ng * nl){
            int ic = id / ng;
            float3 dx;
            float r_sq = 0.0;
            dx.x = p.x - C[ic].x;
            dx.y = p.y - C[ic].y;
            dx.z = p.z - C[ic].z;
            r_sq = dx.x * dx.x + dx.y * dx.y + dx.z * dx.z;
            float F_mag = -(p.w * C[ic].w) / powf(r_sq + epsilon*epsilon, 1.5);
            f.x += F_mag * dx.x;
            f.y += F_mag * dx.y;
            f.z += F_mag * dx.z;
        }
        // Particle-Particle
        else if(id < ng * (nl + ng)){
            int ipp = id / ng - nl;
            float3 dx;
            float r_sq = 0.0;
            dx.x = p.x - P[ipp].x;
            dx.y = p.y - P[ipp].y;
            dx.z = p.z - P[ipp].z;
            r_sq = dx.x * dx.x + dx.y * dx.y + dx.z * dx.z;
            float F_mag = -(p.w * P[ipp].w) / powf(r_sq + epsilon*epsilon, 1.5);
            f.x += F_mag * dx.x;
            f.y += F_mag * dx.y;
            f.z += F_mag * dx.z;
        }
        id += blockDim.x * gridDim.x;
    }
    atomicAdd(&F[ip].x, f.x);
    atomicAdd(&F[ip].y, f.y);
    atomicAdd(&F[ip].z, f.z);
    __syncthreads();
}

extern "C" void Particle_Cell_Force_gpu(Coord4* P, int ng, Coord4* C, int nl, Coord3* F, double epsilon){
    float4 *d_P, *d_C;
    float3 *d_F;

    // Allocate device memory
    hipMalloc((void**)&d_P, sizeof(float4) * ng);
    hipMalloc((void**)&d_C, sizeof(float4) * nl);
    hipMalloc((void**)&d_F, sizeof(float3) * ng);

    // Transfer data from host to device memory
    hipMemcpy(d_P, P, sizeof(float4) * ng, hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, sizeof(float4) * nl, hipMemcpyHostToDevice);

    // Initialize d_F as 0.0 (https://forums.developer.nvidia.com/t/can-i-set-a-floats-to-zero-with-cudamemset/153706)
    hipMemset(d_F, 0, sizeof(float3) * ng);

    // Executing kernel
    int threadsPerBlock = 32;
    int blocksPerGrid = ng;//(ng * (nl + ng) + threadsPerBlock - 1) / threadsPerBlock; // ceil(ng * (nl + ng) / threadsPerBlock)
    Particle_Cell_Kernel<<<blocksPerGrid, threadsPerBlock>>>(d_P, ng, d_C, nl, d_F, (float)epsilon);
    hipDeviceSynchronize();

    // Transfer data back to host memory
    hipMemcpy(F, d_F, sizeof(float3) * ng, hipMemcpyDeviceToHost);

    // Deallocate device memory
    hipFree(d_P);
    hipFree(d_C);
    hipFree(d_F);

    return;
}
